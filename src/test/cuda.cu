#include "hip/hip_runtime.h"
/*
 *
 *                This source code is part of
 *                    ******************
 *                    ***   Pteros   ***
 *                    ******************
 *                 molecular modeling library
 *
 * Copyright (c) 2009-2013, Semen Yesylevskyy
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of Artistic License:
 *
 * Please note, that Artistic License is slightly more restrictive
 * then GPL license in terms of distributing the modified versions
 * of this software (they should be approved first).
 * Read http://www.opensource.org/licenses/artistic-license-2.0.php
 * for details. Such license fits scientific software better then
 * GPL because it prevents the distribution of bugged derivatives.
 *
*/
/*
#include <stdio.h>
#include "cuda_pteros_adaptor.h"
#include <iostream>

const int N = 500000;
const int Nrun = 50000;

int main(void)
{

  float v[N*3];
  for(int i=0;i<N*3;++i) v[i]=i;

  float shift[3] = {1,2,3};

  // GPU
  GPU_Frame f(v,N);
  for(int c=0;c<Nrun;++c){
    f.translate(shift);
  }
  f.get();


 //hipDeviceSynchronize();

  //for(int i=0;i<N*3;++i) std::cout << v[i] << " " << std::endl;

  return 0;
}
*/

#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#define N 1000000
int main(void)
{
    float time_cpu;
    float time_gpu;
    int *a = new int[N];
    int *b = new int[N];
    int *c = new int[N];
    for(int i=0;i<N;i++)
    {
        a[i]=i;
        b[i]=i*i;
    }
    clock_t start_cpu,stop_cpu;
    start_cpu=clock();
    for(int i=0;i<N;i++)
    {
        c[i]=a[i]+b[i];
    }
    stop_cpu=clock();
    time_cpu=(double)(stop_cpu-start_cpu)/CLOCKS_PER_SEC;
    std::cout<<"Time to generate (CPU):"<<time_cpu<<std::endl;



    thrust::device_vector<int> X(N);
    thrust::device_vector<int> Y(N);
    thrust::device_vector<int> Z(N);
    for(int i=0;i<N;i++)
    {
        X[i]=i;
        Y[i]=i*i;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    thrust::transform(X.begin(), X.end(),
        Y.begin(),
        Z.begin(),
        thrust::plus<int>());

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    std::cout<<"Time to generate (thrust):"<<elapsedTime<<std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
